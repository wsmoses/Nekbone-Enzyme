#define LX1 10
#define LY1 10
#define LZ1 10

#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void dax_cuda2_kernel_org(double *__restrict__ w, double *__restrict__ u, double *__restrict__ gxyz, double *__restrict__ dxm1, double *__restrict__ dxtm1){
  //printf("real\n");
/*      real, intent(out) :: w(lx1,ly1,lz1,lelt)
      real u(lx1,ly1,lz1,lelt)
      real ur  (lx1,ly1,lz1,lelt)
      real us  (lx1,ly1,lz1,lelt)
      real ut  (lx1,ly1,lz1,lelt)

      real gxyz(lx1,ly1,lz1,2*ldim,lelt)

      real, intent(in) :: dxm1(lx1,lx1)
      real, intent(in) :: dxtm1(lx1,lx1)*/

  double rtmp,stmp,ttmp,wijke;
      //__shared__ double shdxm1[LX1*LY1];
      #define shdxm1 dxm1
      #define shdtxm1 dxtm1
      //__shared__ double shdtxm1[LX1*LY1];
      //__shared__ double shu[LX1*LY1*LZ1];
      #define shu u
      __shared__ double shur[LX1*LY1*LZ1];
      __shared__ double shus[LX1*LY1*LZ1];
      __shared__ double shut[LX1*LY1*LZ1];
      int l,e,i,j,k,ijk,jk,iii,n,nchunks;

      e = blockIdx.x;
      iii = threadIdx.x;

// Figure out how many strided accesses that this block needs to perform
      nchunks = (LX1*LY1*LZ1-1)/128+1;

      if (iii<LX1*LY1){
        shdxm1[iii] = dxm1[iii];
	//        shdtxm1[iii] = dxtm1[iii];
      }
      i = iii;
      while (i<LX1*LY1*LZ1){
        shu[i] = u[i+e*LX1*LY1*LZ1];
        i = i+128;
      }

// Perform the strided accesses.  Each thread in the block proceeds in
// lockstep.
      __syncthreads();

      if (iii<LX1*LY1){
        shdtxm1[iii] = dxtm1[iii];
      }

      for (n=0; n<nchunks; n++){
        ijk = iii+n*128;
        jk = ijk/LX1;
        i = ijk-jk*LX1;
        k = jk/LY1;
        j = jk-k*LY1;
        if (i<LX1 && j<LY1 && k<LZ1){
	        rtmp = 0.0;
          stmp = 0.0;
          ttmp = 0.0;
          #pragma unroll
          for (l = 0; l<LX1; l++){
            rtmp = rtmp + shdxm1[i+l*LX1] * shu[l+j*LX1+k*LX1*LY1];
            stmp = stmp + shdxm1[j+l*LX1] * shu[i+l*LX1+k*LX1*LY1];
            ttmp = ttmp + shdxm1[k+l*LX1] * shu[i+j*LX1+l*LX1*LY1];
          }
          shur[ijk] = gxyz[ijk+0*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*rtmp
                    + gxyz[ijk+1*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*stmp
                    + gxyz[ijk+2*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*ttmp;
          shus[ijk] = gxyz[ijk+3*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*stmp
                    + gxyz[ijk+2*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*rtmp
                    + gxyz[ijk+4*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*ttmp;
          shut[ijk] = gxyz[ijk+5*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*ttmp
                    + gxyz[ijk+2*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*rtmp
                    + gxyz[ijk+4*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*stmp;
        }
      }

      __syncthreads();

      for (n=0; n<nchunks; n++){
        ijk = iii+n*128;
        jk = ijk/LX1;
        i = ijk-jk*LX1;
        k = jk/LY1; 
        j = jk-k*LY1;
        if (i<LX1 && j<LY1 && k<LZ1){
          wijke = 0.0;
          for (l = 0; l<LX1; l++){
            wijke = wijke + shdtxm1[i+l*LX1] * shur[l+j*LX1+k*LX1*LY1]
                          + shdtxm1[j+l*LX1] * shus[i+l*LX1+k*LX1*LY1]
                          + shdtxm1[k+l*LX1] * shut[i+j*LX1+l*LX1*LY1];	    
          }
          w[ijk+e*LX1*LY1*LZ1] = wijke;
        }
      }
      /*
      for (n=0; n<nchunks; n++){
        ijk = iii+n*128;
        jk = ijk/LX1;
        i = ijk-jk*LX1;
        k = jk/LY1; 
        j = jk-k*LY1;
        if (i<LX1 && j<LY1 && k<LZ1){
	  w[ijk+e*LX1*LY1*LZ1] = w[ijk+e*LX1*LY1*LZ1] + 
	    helm2[ijk+e*LX1*LY1*LZ1] * bm1[ijk+e*LX1*LY1*LZ1] * oshu[ijk];//+e*LX1*LY1*LZ1];
	}
      }
*/
}
#undef shu
#undef shdxm1
#undef shdtxm1

__global__ void ax_cuda2_kernel_org(double *w, double *u, double *gxyz, double *dxm1, double *dxtm1){
  dax_cuda2_kernel_org(w, u, gxyz, dxm1, dxtm1);
}

template<typename... Args>
__device__ void __enzyme_autodiff(void*, Args...);

__global__ void gradient_ax_cuda2_kernel_org(double *w, double *dw, double *u, double *du, double *gxyz, double *dgxyz, double *dxm1, double *ddxm1, double *dxtm1, double *ddxtm1){
  __enzyme_autodiff((void*)dax_cuda2_kernel_org, w, dw, u, du, gxyz, dgxyz, dxm1, ddxm1, dxtm1, ddxtm1);
}

#if 0
// Basic 2d version
__global__ void ax_cuda2_kernel_2d(double *w, double *u, double *gxyz, double *dxm1, double *dxtm1){
  printf("imag\n");
/*      real, intent(out) :: w(lx1,ly1,lz1,lelt)
      real u(lx1,ly1,lz1,lelt)
      real ur  (lx1,ly1,lz1,lelt)
      real us  (lx1,ly1,lz1,lelt)
      real ut  (lx1,ly1,lz1,lelt)

      real gxyz(lx1,ly1,lz1,2*ldim,lelt)

      real, intent(in) :: dxm1(lx1,lx1)
      real, intent(in) :: dxtm1(lx1,lx1)*/

  double rtmp,stmp,ttmp,wijke;
      __shared__ double shdxm1[LX1*LY1];
      __shared__ double shdtxm1[LX1*LY1];
      __shared__ double shu[LX1*LY1*LZ1];
      __shared__ double shur[LX1*LY1*LZ1];
      __shared__ double shus[LX1*LY1*LZ1];
      __shared__ double shut[LX1*LY1*LZ1];
      int l,e,i,j,k,ijk,ij;

      e = blockIdx.x;
      j = threadIdx.y;
      i = threadIdx.x;
      ij = i + j*LX1;
// Figure out how many strided accesses that this block needs to perform

      shdxm1[ij] = dxm1[ij];
      shdtxm1[ij] = dxtm1[ij];
      for( k = 0; k < LZ1; ++k){
        ijk = ij + k*LX1*LY1;
        shu[ijk] = u[ijk + e*LX1*LY1*LZ1];
      }

// Perform the strided accesses.  Each thread in the block proceeds in
// lockstep.
      __syncthreads();

      for (k=0; k<LZ1; ++k){
        ijk = ij + k*LX1*LY1;
        rtmp = 0.0;
        stmp = 0.0;
        ttmp = 0.0;
        for (l = 0; l<LX1; l++){
          rtmp = rtmp + shdxm1[i+l*LX1] * shu[l+j*LX1+k*LX1*LY1];
          stmp = stmp + shdxm1[j+l*LX1] * shu[i+l*LX1+k*LX1*LY1];
          ttmp = ttmp + shdxm1[k+l*LX1] * shu[i+j*LX1+l*LX1*LY1];
        }
        shur[ijk] = gxyz[ijk+0*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*rtmp
                  + gxyz[ijk+1*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*stmp
                  + gxyz[ijk+2*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*ttmp;
        shus[ijk] = gxyz[ijk+3*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*stmp
                  + gxyz[ijk+2*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*rtmp
                  + gxyz[ijk+4*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*ttmp;
        shut[ijk] = gxyz[ijk+5*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*ttmp
                  + gxyz[ijk+2*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*rtmp
                  + gxyz[ijk+4*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*stmp;
      }

      __syncthreads();

      for (k=0; k<LZ1; ++k){
        ijk = ij + k*LX1*LY1;
        wijke = 0.0;
        for (l = 0; l<LX1; l++){
          wijke = wijke + shdtxm1[i+l*LX1] * shur[l+j*LX1+k*LX1*LY1]
                        + shdtxm1[j+l*LX1] * shus[i+l*LX1+k*LX1*LY1]
                        + shdtxm1[k+l*LX1] * shut[i+j*LX1+l*LX1*LY1];	    
        }
        w[ijk+e*LX1*LY1*LZ1] = wijke;
      }
}
__global__ void ax_cuda2_kernel(double* __restrict__ w, const double* __restrict__ u, const double* __restrict__ gxyz, const double* __restrict__ dxm1, const double* __restrict__ dxtm1){
  printf("comp\n");
/*      real, intent(out) :: w(lx1,ly1,lz1,lelt)
      real u(lx1,ly1,lz1,lelt)
      real ur  (lx1,ly1,lz1,lelt)
      real us  (lx1,ly1,lz1,lelt)
      real ut  (lx1,ly1,lz1,lelt)

      real gxyz(lx1,ly1,lz1,2*ldim,lelt)

      real, intent(in) :: dxm1(lx1,lx1)
      real, intent(in) :: dxtm1(lx1,lx1)*/

      double rtmp,stmp,ttmp,wijke;
      __shared__ double shdxm1[LX1*LY1];
      __shared__ double shu[LX1*LY1];
      __shared__ double shur[LX1*LY1];
      __shared__ double shus[LX1*LY1];
      double ru[LZ1];
      double rw[LZ1];
      double rut;
      double G00,G01,G02,G11,G12,G22;
      int l,e,i,j,k,ijk,ij,ele;

      e = blockIdx.x;
      j = threadIdx.y;
      i = threadIdx.x;
      ij = i + j*LX1;
      ele = e*LX1*LY1*LZ1;

      shdxm1[ij] = dxm1[ij];
      #pragma unroll
      for( k = 0; k < LZ1; ++k){
        ru[k] = u[ij + k*LX1*LY1 + ele];
        rw[k] = 0.0;
      }

// Perform the strided accesses.  Each thread in the block proceeds in
// lockstep.
      __syncthreads();
      #pragma unroll
      for (k=0; k<LZ1; ++k){
        ijk = ij + k*LX1*LY1; 
        G00 = gxyz[ijk+0*LX1*LY1*LZ1+ele*6];
        G01 = gxyz[ijk+1*LX1*LY1*LZ1+ele*6];
        G02 = gxyz[ijk+2*LX1*LY1*LZ1+ele*6]; 
        G11 = gxyz[ijk+3*LX1*LY1*LZ1+ele*6];
        G12 = gxyz[ijk+4*LX1*LY1*LZ1+ele*6];
        G22 = gxyz[ijk+5*LX1*LY1*LZ1+ele*6];
        ttmp = 0.0;
        shu[ij] = ru[k];
        for (l = 0; l<LX1; l++){
          ttmp += shdxm1[k+l*LX1] * ru[l];
        }
        __syncthreads();
 
        rtmp = 0.0;
        stmp = 0.0;
        #pragma unroll
        for (l = 0; l<LX1; l++){
          rtmp += shdxm1[i+l*LX1] * shu[l+j*LX1];
          stmp += shdxm1[j+l*LX1] * shu[i+l*LX1];
        }
        shur[ij] = G00*rtmp
                 + G01*stmp
                 + G02*ttmp;
        rut      = G02*rtmp
                 + G12*stmp 
                 + G22*ttmp;
        shus[ij] = G01*rtmp
                 + G11*stmp
                 + G12*ttmp;

      __syncthreads();

        wijke = 0.0;
        #pragma unroll
        for (l = 0; l<LX1; l++){
          wijke += shdxm1[l + i*LX1] * shur[l+j*LX1];
          rw[l] += shdxm1[k+l*LX1] * rut; 
          wijke += shdxm1[l + j*LX1] * shus[i+l*LX1];
        }
        rw[k] += wijke;
      }
      #pragma unroll
      for (k=0; k<LZ1; ++k){
        w[ij + k*LX1*LY1 + ele] = rw[k]; 
      }
}
#endif

extern "C" {
  void ax_cuda2_(double* __restrict__ w, double* __restrict__ u, double* __restrict__ gxyz,
  double* __restrict__ dxm1, double* __restrict__ dxtm1, int *nel){
   printf("calling ax_cuda2_\n");
    //ax_cuda2_kernel<<<*nel,dim3(LX1,LY1,1)>>>(w, u, gxyz, dxm1, dxtm1);
    double dw[1];
    double du[1];
    double dgxyz[1];
    double ddxm1[1];
    double ddxtm1[1];
    gradient_ax_cuda2_kernel_org<<<*nel,dim3(LX1,LY1,1)>>>(w, dw, u, du, gxyz, dgxyz, dxm1, ddxm1, dxtm1, ddxtm1);
 }

  void bandwidth_test_(void*  w, void * u, void* gxyz,
 void* dxm1, void* dxtm1, const int *nel){
    int n = *nel*LX1*LX1*LX1;
    hipMemcpy(u,w,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(u,w,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(u,w,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(gxyz,gxyz,6*n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(gxyz,gxyz,6*n*sizeof(double),hipMemcpyDeviceToDevice);
    //cudaMemcpy(gxyz,gxyz,6*n*sizeof(double),cudaMemcpyDeviceToDevice);
    //cudaMemcpy(gxyz,gxyz,6*n*sizeof(double),cudaMemcpyDeviceToDevice);
    //cudaMemcpy(gxyz,gxyz,6*n*sizeof(double),cudaMemcpyDeviceToDevice);
    //cudaMemcpy(dxm1,dxtm1,LX1*LX1*LX1*sizeof(double),cudaMemcpyDeviceToDevice);
}
  void bandwidth_test2_(void*  w,void* x, void * p, void* z,void* c, void* r, void* cmask, void* gxyz, const int *nel){
    int n = *nel*LX1*LX1*LX1;
    
    hipMemcpy(z,z,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(r,r,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(c,c,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(z,z,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(p,p,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(z,z,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(r,r,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(p,p,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(gxyz,gxyz,6*n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(r,r,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(w,w,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(r,r,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(p,p,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(r,r,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(p,p,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(c,c,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(r,r,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(x,x,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(w,w,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(p,p,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(r,r,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(x,x,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(w,w,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(c,c,n*sizeof(double),hipMemcpyDeviceToDevice);
    hipMemcpy(r,r,n*sizeof(double),hipMemcpyDeviceToDevice);
}
}

