#define LX1 10
#define LY1 10
#define LZ1 10

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void ax_cuda2_kernel_org(double *w, double *u, double *gxyz, double *dxm1, double *dxtm1){
/*      real, intent(out) :: w(lx1,ly1,lz1,lelt)
      real u(lx1,ly1,lz1,lelt)
      real ur  (lx1,ly1,lz1,lelt)
      real us  (lx1,ly1,lz1,lelt)
      real ut  (lx1,ly1,lz1,lelt)

      real gxyz(lx1,ly1,lz1,2*ldim,lelt)

      real, intent(in) :: dxm1(lx1,lx1)
      real, intent(in) :: dxtm1(lx1,lx1)*/

  double rtmp,stmp,ttmp,wijke;
      __shared__ double shdxm1[LX1*LY1];
      __shared__ double shdtxm1[LX1*LY1];
      __shared__ double shu[LX1*LY1*LZ1];
      __shared__ double shur[LX1*LY1*LZ1];
      __shared__ double shus[LX1*LY1*LZ1];
      __shared__ double shut[LX1*LY1*LZ1];
      int l,e,i,j,k,ijk,jk,iii,n,nchunks;

      e = blockIdx.x;
      iii = threadIdx.x;

// Figure out how many strided accesses that this block needs to perform
      nchunks = (LX1*LY1*LZ1-1)/128+1;

      if (iii<LX1*LY1){
        shdxm1[iii] = dxm1[iii];
	//        shdtxm1[iii] = dxtm1[iii];
      }
      i = iii;
      while (i<LX1*LY1*LZ1){
        shu[i] = u[i+e*LX1*LY1*LZ1];
        i = i+128;
      }

// Perform the strided accesses.  Each thread in the block proceeds in
// lockstep.
      __syncthreads();

      if (iii<LX1*LY1){
        shdtxm1[iii] = dxtm1[iii];
      }

      for (n=0; n<nchunks; n++){
        ijk = iii+n*128;
        jk = ijk/LX1;
        i = ijk-jk*LX1;
        k = jk/LY1;
        j = jk-k*LY1;
        if (i<LX1 && j<LY1 && k<LZ1){
	  rtmp = 0.0;
          stmp = 0.0;
          ttmp = 0.0;
          for (l = 0; l<LX1; l++){
            rtmp = rtmp + shdxm1[i+l*LX1] * shu[l+j*LX1+k*LX1*LY1];
            stmp = stmp + shdxm1[j+l*LX1] * shu[i+l*LX1+k*LX1*LY1];
            ttmp = ttmp + shdxm1[k+l*LX1] * shu[i+j*LX1+l*LX1*LY1];
          }
          shur[ijk] = gxyz[ijk+0*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*rtmp
                    + gxyz[ijk+1*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*stmp
                    + gxyz[ijk+2*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*ttmp;
          shus[ijk] = gxyz[ijk+3*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*stmp
                    + gxyz[ijk+2*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*rtmp
                    + gxyz[ijk+4*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*ttmp;
          shut[ijk] = gxyz[ijk+5*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*ttmp
                    + gxyz[ijk+2*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*rtmp
                    + gxyz[ijk+4*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*stmp;
        }
      }

      __syncthreads();

      for (n=0; n<nchunks; n++){
        ijk = iii+n*128;
        jk = ijk/LX1;
        i = ijk-jk*LX1;
        k = jk/LY1; 
        j = jk-k*LY1;
        if (i<LX1 && j<LY1 && k<LZ1){
          wijke = 0.0;
          for (l = 0; l<LX1; l++){
            wijke = wijke + shdtxm1[i+l*LX1] * shur[l+j*LX1+k*LX1*LY1]
                          + shdtxm1[j+l*LX1] * shus[i+l*LX1+k*LX1*LY1]
                          + shdtxm1[k+l*LX1] * shut[i+j*LX1+l*LX1*LY1];	    
          }
          w[ijk+e*LX1*LY1*LZ1] = wijke;
        }
      }
      /*
      for (n=0; n<nchunks; n++){
        ijk = iii+n*128;
        jk = ijk/LX1;
        i = ijk-jk*LX1;
        k = jk/LY1; 
        j = jk-k*LY1;
        if (i<LX1 && j<LY1 && k<LZ1){
	  w[ijk+e*LX1*LY1*LZ1] = w[ijk+e*LX1*LY1*LZ1] + 
	    helm2[ijk+e*LX1*LY1*LZ1] * bm1[ijk+e*LX1*LY1*LZ1] * oshu[ijk];//+e*LX1*LY1*LZ1];
	}
      }
*/
}
// Basic 2d version
__global__ void ax_cuda2_kernel_2d(double *w, double *u, double *gxyz, double *dxm1, double *dxtm1){
/*      real, intent(out) :: w(lx1,ly1,lz1,lelt)
      real u(lx1,ly1,lz1,lelt)
      real ur  (lx1,ly1,lz1,lelt)
      real us  (lx1,ly1,lz1,lelt)
      real ut  (lx1,ly1,lz1,lelt)

      real gxyz(lx1,ly1,lz1,2*ldim,lelt)

      real, intent(in) :: dxm1(lx1,lx1)
      real, intent(in) :: dxtm1(lx1,lx1)*/

  double rtmp,stmp,ttmp,wijke;
      __shared__ double shdxm1[LX1*LY1];
      __shared__ double shdtxm1[LX1*LY1];
      __shared__ double shu[LX1*LY1*LZ1];
      __shared__ double shur[LX1*LY1*LZ1];
      __shared__ double shus[LX1*LY1*LZ1];
      __shared__ double shut[LX1*LY1*LZ1];
      int l,e,i,j,k,ijk,ij;

      e = blockIdx.x;
      j = threadIdx.y;
      i = threadIdx.x;
      ij = i + j*LX1;
// Figure out how many strided accesses that this block needs to perform

      shdxm1[ij] = dxm1[ij];
      shdtxm1[ij] = dxtm1[ij];
      for( k = 0; k < LZ1; ++k){
        ijk = ij + k*LX1*LY1;
        shu[ijk] = u[ijk + e*LX1*LY1*LZ1];
      }

// Perform the strided accesses.  Each thread in the block proceeds in
// lockstep.
      __syncthreads();

      for (k=0; k<LZ1; ++k){
        ijk = ij + k*LX1*LY1;
        rtmp = 0.0;
        stmp = 0.0;
        ttmp = 0.0;
        for (l = 0; l<LX1; l++){
          rtmp = rtmp + shdxm1[i+l*LX1] * shu[l+j*LX1+k*LX1*LY1];
          stmp = stmp + shdxm1[j+l*LX1] * shu[i+l*LX1+k*LX1*LY1];
          ttmp = ttmp + shdxm1[k+l*LX1] * shu[i+j*LX1+l*LX1*LY1];
        }
        shur[ijk] = gxyz[ijk+0*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*rtmp
                  + gxyz[ijk+1*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*stmp
                  + gxyz[ijk+2*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*ttmp;
        shus[ijk] = gxyz[ijk+3*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*stmp
                  + gxyz[ijk+2*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*rtmp
                  + gxyz[ijk+4*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*ttmp;
        shut[ijk] = gxyz[ijk+5*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*ttmp
                  + gxyz[ijk+2*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*rtmp
                  + gxyz[ijk+4*LX1*LY1*LZ1+e*LX1*LY1*LZ1*6]*stmp;
      }

      __syncthreads();

      for (k=0; k<LZ1; ++k){
        ijk = ij + k*LX1*LY1;
        wijke = 0.0;
        for (l = 0; l<LX1; l++){
          wijke = wijke + shdtxm1[i+l*LX1] * shur[l+j*LX1+k*LX1*LY1]
                        + shdtxm1[j+l*LX1] * shus[i+l*LX1+k*LX1*LY1]
                        + shdtxm1[k+l*LX1] * shut[i+j*LX1+l*LX1*LY1];	    
        }
        w[ijk+e*LX1*LY1*LZ1] = wijke;
      }
}
__global__ void ax_cuda2_kernel(double *w, double *u, double *gxyz, double *dxm1, double *dxtm1){
/*      real, intent(out) :: w(lx1,ly1,lz1,lelt)
      real u(lx1,ly1,lz1,lelt)
      real ur  (lx1,ly1,lz1,lelt)
      real us  (lx1,ly1,lz1,lelt)
      real ut  (lx1,ly1,lz1,lelt)

      real gxyz(lx1,ly1,lz1,2*ldim,lelt)

      real, intent(in) :: dxm1(lx1,lx1)
      real, intent(in) :: dxtm1(lx1,lx1)*/

  double rtmp,stmp,ttmp,wijke;
      __shared__ double shdxm1[LX1*LY1];
      __shared__ double shdtxm1[LX1*LY1];
      __shared__ double shu[LX1*LY1];
      __shared__ double shur[LX1*LY1];
      __shared__ double shus[LX1*LY1];
      double ru[LZ1];
      double rw[LZ1];
      double rut;
      double G00,G01,G02,G11,G12,G22;
      int l,e,i,j,k,ijk,ij,ele;

      e = blockIdx.x;
      j = threadIdx.y;
      i = threadIdx.x;
      ij = i + j*LX1;
      ele = e*LX1*LY1*LZ1;

      shdxm1[ij] = dxm1[ij];
      shdtxm1[ij] = dxtm1[ij];
      for( k = 0; k < LZ1; ++k){
        ru[k] = u[ij + k*LX1*LY1 + ele];
        rw[k] = 0.0;
      }

// Perform the strided accesses.  Each thread in the block proceeds in
// lockstep.
      __syncthreads();
      #pragma unroll
      for (k=0; k<LZ1; ++k){
        ijk = ij + k*LX1*LY1; 
        G00 = gxyz[ijk+0*LX1*LY1*LZ1+ele*6];
        G01 = gxyz[ijk+1*LX1*LY1*LZ1+ele*6];
        G02 = gxyz[ijk+2*LX1*LY1*LZ1+ele*6]; 
        G11 = gxyz[ijk+3*LX1*LY1*LZ1+ele*6];
        G12 = gxyz[ijk+4*LX1*LY1*LZ1+ele*6];
        G22 = gxyz[ijk+5*LX1*LY1*LZ1+ele*6];
 
        ttmp = 0.0;
        shu[ij] = ru[k];
        for (l = 0; l<LX1; l++){
          ttmp += shdtxm1[l + k*LX1] * ru[l];
        }
        __syncthreads();
 
        rtmp = 0.0;
        stmp = 0.0;
        #pragma unroll
        for (l = 0; l<LX1; l++){
          rtmp += shdxm1[i+l*LX1] * shu[l+j*LX1];
          stmp += shdxm1[j+l*LX1] * shu[i+l*LX1];
        }
        shur[ij] = G00*rtmp
                 + G01*stmp
                 + G02*ttmp;
        shus[ij] = G01*rtmp
                 + G11*stmp
                 + G12*ttmp;
        rut      = G02*rtmp
                 + G12*stmp 
                 + G22*ttmp;

      __syncthreads();

        wijke = 0.0;
        #pragma unroll
        for (l = 0; l<LX1; l++){
          wijke += shdxm1[l + i*LX1] * shur[l+j*LX1]
                        + shdxm1[l + j*LX1] * shus[i+l*LX1];
          rw[l] += shdtxm1[l+k*LX1] * rut; 
        }
        rw[k] += wijke;
      }
      #pragma unroll
      for (k=0; k<LZ1; ++k){
        w[ij + k*LX1*LY1 + ele] = rw[k]; 
      }
}

extern "C" {
  void ax_cuda2_(double *w, double *u, double *gxyz, double *dxm1, double *dxtm1,  int *nel){
    ax_cuda2_kernel<<<*nel,dim3(LX1,LY1,1)>>>(w, u, gxyz, dxm1, dxtm1);
}
}